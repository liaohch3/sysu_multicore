#include "hip/hip_runtime.h"
#include "core.h"
#define NUMLEN 16

__global__ void kernel(int width, int height, float *input, float *output) {
    for(int i = 0; i < width; i++){
        for(int j = 0; j < height; j++){

            int sum = 0;
            int counts[NUMLEN] = {0};
            for(int m = i-2; m <= i+2; m++){
                for(int n = j-2; n <= j+2; n++){
                    if(0 <= m && m < height && 0 <= n && n < width){
                        int num = input[m*width+n];
                        counts[num]++;
                        sum++;
                    }
                }
            }

            double res = logf(sum);
            for(int m = 0; m < NUMLEN; m++){
                int count = counts[m];
                if(count != 0){
                    res -= count * logf(count) / sum;
                }
            }

            output[i*width+j] = res;
        }
    }
}

void cudaCallback(int width, int height, float *sample, float **result) {
    int size = width * height;
    float *input_d, *output_d;

    // Allocate device memory and copy data from host to device
    CHECK(hipMalloc((void **)&input_d, sizeof(float)*size));
    CHECK(hipMalloc((void **)&output_d, sizeof(float)*size));
    CHECK(hipMemcpy(input_d, sample, sizeof(float)*size, hipMemcpyHostToDevice));

    // Invoke the device function
    kernel<<< 1, 1 >>>(width, height, input_d, output_d);
    hipDeviceSynchronize();

    // Copy back the results and de-allocate the device memory
    *result = (float *)malloc(sizeof(float)*size);
    CHECK(hipMemcpy(*result, output_d, sizeof(float)*size, hipMemcpyDeviceToHost));
    CHECK(hipFree(input_d));
    CHECK(hipFree(output_d));

    // Note that you don't have to free sample and *result by yourself
}